#include "hip/hip_runtime.h"

struct Params
{
  uchar4*  image;
  unsigned int  image_width;
  unsigned int  image_height;
  float3   cam_eye;
  float3   cam_u, cam_v, cam_w;
  OptixTraversableHandle handle;
};

extern "C"
{
  __constant__ Params params;
}

extern "C" __global__ void __raygen__rg()
{
  // Lookup our location within the launch grid
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();

  // Map our launch idx to a screen location and create a ray from 
  // the camera location through the screen
  float3 ray_origin, ray_direction;
  computeRay( idx, dim, ray_origin, ray_direction );

  // Trace the ray against our scene hierarchy
  unsigned int p0, p1, p2;
  optixTrace(
    params.handle,
    ray_origin,
    ray_direction,
    0.0f,   // Min intersection distance
    1e16f,  // Max intersection distance
    0.0f,   // ray-time -- used for motion blur
    OptixVisibilityMask(255), // Specify always visible
    OPTIX_RAY_FLAG_NONE,
    0,      // SBT offset -- See SBT discussion
    0,      // SBT stride -- See SBT discussion 
    0,      // missSBTIndex -- See SBT discussion
    p0, p1, p2 ); // These 32b values are the ray payload

  // Our results were packed into opaque 32b registers
  float3 result;
  result.x = int_as_float(p0);
  result.y = int_as_float(p1);
  result.z = int_as_float(p2);
  // Record results in our output raster
  params.image[idx.y * params.image_width + idx.x] = make_color( result );
}
