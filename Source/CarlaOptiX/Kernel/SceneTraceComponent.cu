#include "hip/hip_runtime.h"
#include <optix.h>



struct KernelArgs
{
	OptixTraversableHandle gas;
	float3* result_buffer;
	float3 origin;
	float3 direction;
	uint2 viewport_size;
	float ray_time;
	float epsilon;
	float max_distance;
};

struct RayContext
{
	float3 hit_position;
	float3 hit_tangent;
	float3 hit_normal;
};



extern "C" __constant__ KernelArgs args;



#define MAKE_POINTER(low, high)			(void*)((uintptr_t)(low) | ((uintptr_t)(high) << 32))
#define POINTER_LOW32(ptr) (uint32_t)((uintptr_t)(ptr) & UINT32_MAX)
#define POINTER_HIGH32(ptr) (uint32_t)(((uintptr_t)(ptr) >> 32) & UINT32_MAX)



extern "C" __global__ void __raygen__main()
{
	const uint3 launch_id = optixGetLaunchIndex();
	const uint32_t sbt_offset = 0;
	const uint32_t sbt_stride = 0;
	const uint32_t sbt_miss_index = 0;

	RayContext ray_ctx;
	auto low32 = POINTER_LOW32(&ray_ctx);
	auto high32 = POINTER_HIGH32(&ray_ctx);

	optixTrace(
		args.gas,
		args.origin,
		args.direction,
		args.epsilon,
		args.max_distance,
		OptixVisibilityMask(0xFF),
		0,
		sbt_offset,
		sbt_stride,
		sbt_miss_index,
		low32,
		high32);
}

extern "C" __global__ void __anyhit__main()
{
	const uint3 launch_id = optixGetLaunchIndex();
}

extern "C" __global__ void __closesthit__main()
{
	const uint3 launch_id = optixGetLaunchIndex();
	auto& ray_ctx = *(RayContext*)MAKE_POINTER(
		optixGetPayload_0(),
		optixGetPayload_1());
}

extern "C" __global__ void __miss__main()
{
	const uint3 launch_id = optixGetLaunchIndex();
	auto& ray_ctx = *(RayContext*)MAKE_POINTER(
		optixGetPayload_0(),
		optixGetPayload_1());
}

extern "C" __global__ void __intersection__main()
{
	const uint3 launch_id = optixGetLaunchIndex();
	auto& ray_ctx = *(RayContext*)MAKE_POINTER(
		optixGetPayload_0(),
		optixGetPayload_1());
}