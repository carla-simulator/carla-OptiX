#include "hip/hip_runtime.h"



extern "C" __global__ void __closesthit__ch()
{
  // When built-in triangle intersection is used, a number of fundamental 
  // attributes are provided by the OptiX API, including barycentric 
  // coordinates.
  const float2 barycentrics = optixGetTriangleBarycentrics();
  // Convert to color and assign to our payload outputs.
  const float3 c = make_float3(barycentrics, 1.0f); 
  optixSetPayload_0(float_as_int(c.x));
  optixSetPayload_1(float_as_int(c.y));
  optixSetPayload_2(float_as_int(c.z));
}
